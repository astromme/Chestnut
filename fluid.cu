#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// Prototypes
__global__ void grow();

float** create2DFloatArray(int rows, int cols) {
  float** theArray;
  theArray = (float**) malloc(cols*sizeof(float*));
  for (int i = 0; i < cols; i++)
    theArray[i] = (double*) malloc(rows*sizeof(double));
  return theArray;
} 

// Host function
int
main(int argc, char** argv)
{

  // 400x400 array
  float** localSource = create2DFloatArray(400, 400);
  float** localDest = create2DFloatArray(400, 400);

  // allocate memory on the device 
  char *d_str;
  size_t size = sizeof(str);
  hipMalloc((void**)&d_str, size);

  // copy the string to the device
  hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

  // set the grid and block sizes
  dim3 dimGrid(2);   // one block per word  
  dim3 dimBlock(6); // one thread per character
  
  // invoke the kernel
  helloWorld<<< dimGrid, dimBlock >>>(d_str);

  // retrieve the results from the device
  hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

  // free up the allocated memory on the device
  hipFree(d_str);
  
  // everyone's favorite part
  printf("%s\n", str);

  return 0;
}

// Device kernel
__global__ void
helloWorld(char* str)
{
  // determine where in the thread grid we are
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // unmangle output
  str[idx] += idx;
}
