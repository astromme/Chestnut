
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define HANDLE_ERROR(a, msg) \
  { \
    hipError_t ret; \
    if ((ret=(a)) != hipSuccess) { \
      printf(msg); \
      printf("error %d\n", ret); \
    } \
  }

// Thread block size
#define BLOCK_SIZE 16
// Forward declaration of the device multiplication function
__global__ void Muld(float*, float*, int, int, float*);
// Host multiplication function
// Compute C = A * B
// hA is the height of A
// wA is the width of A
// wB is the width of B
void Mul(const float* A, const float* B, int hA, int wA, int wB,
				float* C)
{
		int size;
		// Load A and B to the device
		float* Ad;
		size = hA * wA * sizeof(float);
		hipMalloc((void**)&Ad, size);
		hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
		float* Bd;
		size = wA * wB * sizeof(float);
		hipMalloc((void**)&Bd, size);
		hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
		// Allocate C on the device
		float* Cd;
		size = hA * wB * sizeof(float);
		hipMalloc((void**)&Cd, size);
		// Compute the execution configuration assuming
		// the matrix dimensions are multiples of BLOCK_SIZE
		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid(wB / dimBlock.x, hA / dimBlock.y);
		// Launch the device computation
		Muld<<<dimGrid, dimBlock>>>(Ad, Bd, wA, wB, Cd);
		// Read C from the device
		hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
		// Free device memory
		hipFree(Ad);
		hipFree(Bd);
		hipFree(Cd);
}

// Device multiplication function called by Mul()
// Compute C = A * B
// wA is the width of A
// wB is the width of B
__global__ void Muld(float* A, float* B, int wA, int wB, float* C)
{
		// Block index
		int bx = blockIdx.x;
		int by = blockIdx.y;
		// Thread index
		int tx = threadIdx.x;
		int ty = threadIdx.y;
		// Index of the first sub-matrix of A processed by the block
		int aBegin = wA * BLOCK_SIZE * by;
		// Index of the last sub-matrix of A processed by the block
		int aEnd = aBegin + wA - 1;
		// Step size used to iterate through the sub-matrices of A
		int aStep = BLOCK_SIZE;
		// Index of the first sub-matrix of B processed by the block
		int bBegin = BLOCK_SIZE * bx;
		// Step size used to iterate through the sub-matrices of B
		int bStep = BLOCK_SIZE * wB;
		// The element of the block sub-matrix that is computed
		// by the thread
		float Csub = 0;
		// Loop over all the sub-matrices of A and B required to
		// compute the block sub-matrix
		for (int a = aBegin, b = bBegin;
						a <= aEnd;
						a += aStep, b += bStep) {
				// Shared memory for the sub-matrix of A
				__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
				// Shared memory for the sub-matrix of B
				__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
				// Load the matrices from global memory to shared memory;
				// each thread loads one element of each matrix
				As[ty][tx] = A[a + wA * ty + tx];
				Bs[ty][tx] = B[b + wB * ty + tx];
				// Synchronize to make sure the matrices are loaded
				__syncthreads();
				// Multiply the two matrices together;
				// each thread computes one element
				// of the block sub-matrix
				for (int k = 0; k < BLOCK_SIZE; ++k)
						Csub += As[ty][k] * Bs[k][tx];
				// Synchronize to make sure that the preceding
				// computation is done before loading two new
				// sub-matrices of A and B in the next iteration
				__syncthreads();
		}
		// Write the block sub-matrix to global memory;
		// each thread writes one element
		int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
		C[c + wB * ty + tx] = Csub;
}

int main(int argc, char* argv[]) {
		
  float *dev_a;
  float *dev_b;
  float *dev_c;

  /*
  int width_a = 64*BLOCK_SIZE;
  int height_a = 32*BLOCK_SIZE;
  int width_b = height_a;
  int height_b = 48*BLOCK_SIZE;
  int width_c = width_a;
  int height_c = height_b;
  */

  int width_a = 512;
  int height_a = 512;
  int width_b = 512;
  int height_b = 512;
  int width_c = 512;
  int height_c = 512;

  // allocate memory space on GPU
  HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(float)*width_a*height_a),
      "malloc dev_a") ;
  HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(float)*width_b*height_b),
      "malloc dev_b") ;
  HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(float)*width_c*height_c),
      "malloc dev_c") ;

  // Compute the execution configuration assuming
  // the matrix dimensions are multiples of BLOCK_SIZE
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(width_b / dimBlock.x, height_a / dimBlock.y);
  // Launch the device computation
  for (int i=0; i<100; i++) {
		Muld<<<dimGrid, dimBlock>>>(dev_a, dev_b, width_a, width_b, dev_c);
  }
  
  //Mul(dev_a, dev_b, height_a, width_a, width_b, dev_c);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}
