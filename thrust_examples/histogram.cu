#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/adjacent_difference.h>

#include <iostream>
#include <iterator>
#include <cstdlib>

// This example compute the histogram [1] and cumulative
// histogram of an array of integer values.
//
// [1] http://en.wikipedia.org/wiki/Histogram


int main(void)
{
    const size_t N = 30;

    // generate random data on the host
    thrust::host_vector<int> h_data(N);
    for(size_t i = 0; i < N; i++)
        h_data[i] = rand() % 10;

    // transfer data to device
    thrust::device_vector<int> d_data(h_data);
    
    // print the initial data
    std::cout << "initial data" << std::endl;
    thrust::copy(d_data.begin(), d_data.end(), std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;

    // sort data to bring equal elements together
    thrust::sort(d_data.begin(), d_data.end());
    
    // print the sorted data
    std::cout << "sorted data" << std::endl;
    thrust::copy(d_data.begin(), d_data.end(), std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;

    // number of histogram bins is equal to the maximum value plus one
    const int num_bins = d_data.back() + 1;

    // allocate storage for the cumulative histogram and histogram
    thrust::device_vector<int> d_cumulative_histogram(num_bins);
    thrust::device_vector<int> d_histogram(num_bins);
    
    // find the end of each bin of values
    thrust::counting_iterator<int> search_begin(0);
    thrust::upper_bound(d_data.begin(),
                        d_data.end(),
                        search_begin,
                        search_begin + num_bins,
                        d_cumulative_histogram.begin());
    
    // print the cumulative histogram
    std::cout << "cumulative histogram" << std::endl;
    thrust::copy(d_cumulative_histogram.begin(), d_cumulative_histogram.end(), std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;

    // compute the histogram by taking differences of the cumulative histogram
    thrust::adjacent_difference(d_cumulative_histogram.begin(),
                                d_cumulative_histogram.end(),
                                d_histogram.begin());

    // print the histogram
    std::cout << "histogram" << std::endl;
    thrust::copy(d_histogram.begin(), d_histogram.end(), std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;
    
    return 0;
}

