#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include <hip/hip_runtime.h>
#include <cstdlib>

// This examples compares sorting performance using Array of Structures (AoS)
// and Structure of Arrays (SoA) data layout.  Legacy applications will often
// store data in C/C++ structs, such as MyStruct defined below.  Although 
// Thrust can process array of structs, it is typically less efficient than
// the equivalent structure of arrays layout.  In this particular example,
// the optimized SoA approach is approximately *five times faster* than the
// traditional AoS method.  Therefore, it is almost always worthwhile to
// convert AoS data structures to SoA.

struct MyStruct
{
    int key;
    float value;

    bool operator<(const MyStruct other) const
    {
        return key < other.key;
    }
};

void initialize_keys(thrust::device_vector<int>& keys)
{
    thrust::host_vector<int> h_keys(keys.size());

    for(size_t i = 0; i < h_keys.size(); i++)
        h_keys[i] = rand();

    keys = h_keys;
}


void initialize_keys(thrust::device_vector<MyStruct>& structures)
{
    thrust::host_vector<MyStruct> h_structures(structures.size());

    for(size_t i = 0; i < h_structures.size(); i++)
        h_structures[i].key = rand();

    structures = h_structures;
}

int main(void)
{
    size_t N = 1000000;
    hipEvent_t start;
    hipEvent_t end;
    float elapsed_time;
    
    hipEventCreate(&start);
    hipEventCreate(&end);

    // Sort Key-Value pairs using Array of Structures (AoS) storage 
    {
        thrust::device_vector<MyStruct> structures(N);

        initialize_keys(structures);

        hipEventRecord(start,0);

        thrust::sort(structures.begin(), structures.end());

        hipEventSynchronize(end);
        hipEventRecord(end,0);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, start, end);

        std::cout << "AoS sort took " << elapsed_time << " milliseconds" << std::endl;
    }
   
    // Sort Key-Value pairs using Structure of Arrays (SoA) storage 
    {
        thrust::device_vector<int>   keys(N);
        thrust::device_vector<float> values(N);

        initialize_keys(keys);

        hipEventRecord(start,0);

        thrust::sort_by_key(keys.begin(), keys.end(), values.begin());

        hipDeviceSynchronize();
        hipEventRecord(end,0);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, start, end);

        std::cout << "SoA sort took " << elapsed_time << " milliseconds" << std::endl;
    }

    return 0;
}

