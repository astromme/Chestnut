#include <thrust/host_vector.h>
#include <thrust/remove.h>

// This example generates random points in the 
// unit square [0,1)x[0,1) and then removes all 
// points where x^2 + y^2 > 1
//
// The x and y coordinates are stored in separate arrays
// and a zip_iterator is used to combine them together

template <typename T>
struct is_outside_circle
{
    template <typename Tuple>
    bool operator()(const Tuple& tuple) const
    {
        // unpack the tuple into x and y coordinates
        const T x = thrust::get<0>(tuple);
        const T y = thrust::get<1>(tuple);

        if (x*x + y*y > 1)
            return true;
        else
            return false;
    }
};

int main(void)
{
    const size_t N = 20;

    // generate random points in the unit square on the host
    thrust::host_vector<float> x(N);
    thrust::host_vector<float> y(N);
    for(size_t i = 0; i < N; i++)
    {
        x[i] = rand() / (RAND_MAX + 1.0f);
        y[i] = rand() / (RAND_MAX + 1.0f);
    }

    // print the inital points
    std::cout << "Generated " << N << " points" << std::endl;
    for(size_t i = 0; i < N; i++)
        std::cout << "(" << x[i] << "," << y[i] << ")" << std::endl;
    std::cout << std::endl;

    // remove points where x^2 + y^2 > 1 and determine new array sizes
    size_t new_size = thrust::remove_if(thrust::make_zip_iterator(thrust::make_tuple(x.begin(), y.begin())),
                                        thrust::make_zip_iterator(thrust::make_tuple(x.end(), y.end())),
                                        is_outside_circle<float>())
                      - thrust::make_zip_iterator(thrust::make_tuple(x.begin(), y.begin()));

    // resize the vectors (note: this does not free any memory)
    x.resize(new_size);
    y.resize(new_size);

    // print the filtered points
    std::cout << "After stream compaction, " << new_size << " points remain" << std::endl;
    for(size_t i = 0; i < new_size; i++)
        std::cout << "(" << x[i] << "," << y[i] << ")" << std::endl;

    return 0;
}

