// Of course needed for printf()
#include <stdio.h>
// The main cuda include.
#include <hip/hip_runtime.h>
// only needed for KDevelop4 to find the headers/definitions to give
// syntax highlighting and command completion.
#include <hip/hip_runtime_api.h>


// Prototypes
__global__ void grow(float* array);
void printArray(float* array, int rows, int cols);

// Host function. Runs on CPU
int main(int argc, char** argv) {
  int N = 8192; // Size of our square array;
  float *host; // Memory on host (cpu) side
  float *dev; // Memory on device (gpu) side
  
  // Allocate memory on the host that the gpu can access quickly
  hipHostMalloc((void**)&host, N*N*sizeof(float), hipHostMallocDefault); 
  // the above is more or less the same as:
  //host = (float*) malloc(N*N*sizeof(float));
  
  // Allocate memory on the device that is of the same size as our host array
  hipMalloc((void**)&dev, N*N*sizeof(float));
  
  // Initialize our 2d array to all 0s. 
  for(int i=0; i<N*N; i++) {
    host[i] = rand() & 0xFF;
  }
  
  printf("Initial Array:\n");
  printArray(host, 10, 10);
  
  // Copy the array host to dev. We must give it how many bytes to copy hence the sizeof()
  hipMemcpy(dev, host, N*N*sizeof(float), hipMemcpyHostToDevice);

  // run the 'grow' kernel on the device
  // 1 tells it to have one single block
  // dim3(N,N) tells it to have N by N threads in that block.
  // Give the grow kernel the 'dev' array and the number of columns (N)
  grow<<<256, dim3(512, 512)>>>(dev);
  
  // Once the kernel has run, copy back the 'dev' array to the 'host' array
  hipMemcpy(host, dev, N*N*sizeof(float), hipMemcpyDeviceToHost);
  
  // Now it should have incrementing numbers
  printf("After 'grow' kernel:\n");
  printArray(host, 10, 10);
    
  // free up the allocated memory on the device
  hipFree(dev);
  free(host);
  
  return 0;
}


void printArray(float* array, int rows, int cols) {
  for(int row=0; row<rows; row++) {
    for (int col=0; col<cols; col++) {
      // calculate the spot in the 2d array that we want.
      float e = array[(row * cols) + col];
      printf("%.1f\t", e);
    }
    printf("\n");
  }
  printf("\n");
}

__global__ void grow(float* array) {
  int index = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
  array[index] += 10;
//   int row = threadIdx.x;
//   int col = threadIdx.y;
//   // set each slot to its index number
//   array[row*cols + col] = row*cols + col;
}
