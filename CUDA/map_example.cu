// Of course needed for printf()
#include <stdio.h>
// The main cuda include.
#include <hip/hip_runtime.h>
// only needed for KDevelop4 to find the headers/definitions to give
// syntax highlighting and command completion.
#include <hip/hip_runtime_api.h>


// Prototypes
template <typename FCN>
__global__ void map(float* array, int cols, FCN fcn);

float plusThree(float val);
void printArray(float* array, int rows, int cols);

// Host function. Runs on CPU
int main(int argc, char** argv) {
  int N = 8; // Size of our square array;
  float *host; // Memory on host (cpu) side
  float *dev; // Memory on device (gpu) side
  
  // Allocate memory on the host that the gpu can access quickly
  hipHostMalloc((void**)&host, N*N*sizeof(float), hipHostMallocDefault); 
  // the above is more or less the same as:
  //host = (float*) malloc(N*N*sizeof(float));
  
  // Allocate memory on the device that is of the same size as our host array
  hipMalloc((void**)&dev, N*N*sizeof(float));
  
  // Initialize our 2d array to all 0s. 
  for(int i=0; i<N*N; i++) {
    host[i] = 0;
  }
  
  printf("Initial Array\n");
  printArray(host, N, N);
  
  // Copy the array host to dev. We must give it how many bytes to copy hence the sizeof()
  hipMemcpy(dev, host, N*N*sizeof(float), hipMemcpyHostToDevice);

  // run the 'map' kernel on the device
  // 1 tells it to have one single block
  // dim3(N,N) tells it to have N by N threads in that block.
  // Give the grow kernel the 'dev' array and the number of columns (N)
  map<<<1, dim3(N,N)>>>(dev, N, plusThree);
  
  // Once the kernel has run, copy back the 'dev' array to the 'host' array
  hipMemcpy(host, dev, N*N*sizeof(float), hipMemcpyDeviceToHost);
  
  // Now it should have incrementing numbers
  printf("After 'map' kernel:\n");
  printArray(host, N, N);
  
  // free up the allocated memory on the device
  hipFree(dev);
  free(host);
  
  return 0;
}

template <typename FCN> 
__global__ void map(float* array, int cols, FCN fcn) {
  int row = threadIdx.x;
  int col = threadIdx.y;
  
  // set each slot to its index number
  //array[row*cols + col] += 3;
  array[row*cols + col] = fcn(array[row*cols + col]);
}

float plusThree(float val){
  return val + 3;
}

void printArray(float* array, int rows, int cols) {
  for(int row=0; row<rows; row++) {
    for (int col=0; col<cols; col++) {
      // calculate the spot in the 2d array that we want.
      float e = array[(row * cols) + col];
      printf("%.1f\t", e);
    }
    printf("\n");
  }
  printf("\n");
}
