// Of course needed for printf()
#include <stdio.h>
// The main cuda include.
#include <hip/hip_runtime.h>
// only needed for KDevelop4 to find the headers/definitions to give
// syntax highlighting and command completion.
#include <hip/hip_runtime_api.h>


// Prototypes
__global__ void grow(float* array, int cols);
__global__ void shrink(float* array, int cols);
void printArray(float* array, int rows, int cols);

// Host function. Runs on CPU
int main(int argc, char** argv) {
  int N = 8; // Size of our square array;
  float *host; // Memory on host (cpu) side
  float *dev; // Memory on device (gpu) side
  
  // Allocate memory on the host that the gpu can access quickly
  hipHostMalloc((void**)&host, N*N*sizeof(float), hipHostMallocDefault); 
  // the above is more or less the same as:
  //host = (float*) malloc(N*N*sizeof(float));
  
  // Allocate memory on the device that is of the same size as our host array
  hipMalloc((void**)&dev, N*N*sizeof(float));
  
  // Initialize our 2d array to all 0s. 
  for(int i=0; i<N*N; i++) {
    host[i] = 0;
  }
  
  printf("Initial Array\n");
  printArray(host, N, N);
  
  // Copy the array host to dev. We must give it how many bytes to copy hence the sizeof()
  hipMemcpy(dev, host, N*N*sizeof(float), hipMemcpyHostToDevice);

  // run the 'grow' kernel on the device
  // 1 tells it to have one single block
  // dim3(N,N) tells it to have N by N threads in that block.
  // Give the grow kernel the 'dev' array and the number of columns (N)
  grow<<<1, dim3(N,N)>>>(dev, N);
  
  // Once the kernel has run, copy back the 'dev' array to the 'host' array
  hipMemcpy(host, dev, N*N*sizeof(float), hipMemcpyDeviceToHost);
  
  // Now it should have incrementing numbers
  printf("After 'grow' kernel:\n");
  printArray(host, N, N);
  
  // No need to recopy the memory, it's already on the device
  shrink<<<1, dim3(N,N)>>>(dev, N);
  hipMemcpy(host, dev, N*N*sizeof(float), hipMemcpyDeviceToHost);
  
  // Now each row should have the same value for all columns
  printf("After 'shrink' kernel:\n");
  printArray(host, N, N);  
  
  // free up the allocated memory on the device
  hipFree(dev);
  
  return 0;
}


void printArray(float* array, int rows, int cols) {
  for(int row=0; row<rows; row++) {
    for (int col=0; col<cols; col++) {
      // calculate the spot in the 2d array that we want.
      float e = array[(row * cols) + col];
      printf("%.1f\t", e);
    }
    printf("\n");
  }
  printf("\n");
}

__global__ void grow(float* array, int cols) {
  int row = threadIdx.x;
  int col = threadIdx.y;
  // set each slot to its index number
  array[row*cols + col] = row*cols + col;
}

__global__ void shrink(float* array, int cols) {
  int row = threadIdx.x;
  int col = threadIdx.y;
  // shrink each slot by its column number
  array[row*cols + col] -= col;
}